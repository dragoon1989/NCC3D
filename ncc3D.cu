#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <cmath>
#include <fstream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <numeric>

#include "scan3D_kernels.cuh"
#include "reduce3D_kernels.cuh"

// compute Z = conv(X, Y) using FFT
template <typename Layout>
int conv3D_using_FFT(float* dX3D, float* dY3D, float* dZ3D);

// let's test NCC3D computation!
int main(int argc, char* argv[]) {
    // test data type
    using T = float;
    // image volume shape
    constexpr int IM = 128;
    constexpr int IN = 128;
    constexpr int IK = 128;
    // template volume shape
    constexpr int TM = 64;
    constexpr int TN = 64;
    constexpr int TK = 64;
    // output volume shape (must be padded to avoid frequency aliasing)
    constexpr int OM = IM - TM + 1;
    constexpr int ON = IN - TN + 1;
    constexpr int OK = IK - TK + 1;

    // generate image volume
    std::vector<T> image(256 * 256 * 256, T{});
    std::ifstream ins;
    ins.open("input_image-128x128x128.raw", std::ios::binary);
    for (int k = 0; k < IK; k++) {
        for (int j = 0; j < IN; j++) {
            ins.read((char*)(image.data() + j * 256 + k * 256 * 256), sizeof(T) * IM);
        }
    }
    ins.close();

    // generate template volume
    std::vector<T> template_img(256 * 256 * 256, T{});
    ins.open("template_img-64x64x64.raw", std::ios::binary);
    for (int k = 0; k < TK; k++) {
        for (int j = 0; j < TN; j++) {
            ins.read((char*)(template_img.data() + j * 256 + k * 256 * 256), sizeof(T) * TM);
        }
    }
    ins.close();

    // allocate global memory
    hipError_t err = hipSuccess;
    T* dimage, * dimg_sumtable, * energy_sumtable;
    hipMalloc(&dimage, ALIGN128(sizeof(T) * image.size()));
    hipMalloc(&dimg_sumtable, ALIGN128(sizeof(T) * IM * IN * IK));
    hipMalloc(&energy_sumtable, ALIGN128(sizeof(T) * IM * IN * IK));
    hipMemcpy(dimage, image.data(), sizeof(T) * image.size(), hipMemcpyHostToDevice);

    T* d_template_img, * d_sum_workspace, * d_sum2_workspace;
    std::vector<T> h_partital_sum((TM * TN * TK + 16383) / 16384);
    std::vector<T> h_partital_sum2((TM * TN * TK + 16383) / 16384);
    hipMalloc(&d_template_img, ALIGN128(sizeof(T) * 256 * 256 * 256));
    hipMalloc(&d_sum_workspace, ALIGN128(sizeof(T) * ((TM * TN * TK + 16383) / 16384)));
    hipMalloc(&d_sum2_workspace, ALIGN128(sizeof(T) * ((TM * TN * TK + 16383) / 16384)));
    hipMemcpy(d_template_img, template_img.data(), sizeof(T) * template_img.size(), hipMemcpyHostToDevice);

    // generate convolved matrix
    std::vector<T> conv(256 * 256 * 256, T{});
    T* dconv, * doutput;
    hipMalloc(&dconv, ALIGN128(sizeof(T) * conv.size()));
    doutput = dconv;
    hipMemcpy(dconv, conv.data(), sizeof(T) * conv.size(), hipMemcpyHostToDevice);
    if (0 != conv3D_using_FFT<Layout3D<Shape3D<256, 256, 256>>>(dimage, d_template_img, dconv)) {
        return -1;
    }

    // create CUDA streams:
    // compute sumtable --------s1------->|normalization----default stream---->|fetch ncc result
    // compute sumtable2 ------------s2------------>|
    // compute reduce sum ---s3-->|fetch result --->|
    // compute reduce sum2 --s4-->|fetch result --->|
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipStream_t s3, s4;
    hipStreamCreate(&s3);
    hipStreamCreate(&s4);

    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    hipEventRecord(beg, s1);   // this is inaccurate...

    // 1. generate sumtables
    // compute image volume sumtable
    sumtable3D_stage1_kernel<T, 256, 256 * 256, IM, IM* IN> << <IK, dim3(128, 2), 0, s1 >> > (dimage, dimg_sumtable);
    sumtable3D_stage2_kernel<T, IM, IM* IN> << <IK, dim3(128, 2), 0, s1 >> > (dimg_sumtable);
    // compute energy image volume sumtable
    sumtable3D_stage1_kernel<T, 256, 256 * 256, IM, IM* IN, SQR<T>> << <IK, dim3(128, 2), 0, s2 >> > (dimage, energy_sumtable);
    sumtable3D_stage2_kernel<T, IM, IM* IN> << <IK, dim3(128, 2), 0, s2 >> > (energy_sumtable);

    // 2. compute template mean values
    reduce_sum_16384_kernel<T, Layout3D<Shape3D<TM, TN, TK>, Stride3D<1, 256, 256 * 256>>> << <(TM * TN * TK + 16383) / 16384, 1024, 0, s3 >> > (d_template_img, d_sum_workspace);
    reduce_sum_16384_kernel<T, Layout3D<Shape3D<TM, TN, TK>, Stride3D<1, 256, 256 * 256>>, SQR<T>> << <(TM * TN * TK + 16383) / 16384, 1024, 0, s4 >> > (d_template_img, d_sum2_workspace);
    err = hipStreamSynchronize(s3);
    if (hipSuccess != err) {
        printf("Fail to compute template sum, CUDA error %d!\n", err);
        return -1;
    }
    err = hipStreamSynchronize(s4);
    if (hipSuccess != err) {
        printf("Fail to compute template sum, CUDA error %d!\n", err);
        return -1;
    }
    hipMemcpy(h_partital_sum.data(), d_sum_workspace, sizeof(T) * h_partital_sum.size(), hipMemcpyDeviceToHost);
    hipMemcpy(h_partital_sum2.data(), d_sum2_workspace, sizeof(T) * h_partital_sum2.size(), hipMemcpyDeviceToHost);
    T template_img_sum = std::accumulate(h_partital_sum.begin(), h_partital_sum.end(), T{});
    T template_img2_sum = std::accumulate(h_partital_sum2.begin(), h_partital_sum2.end(), T{});

    // 3. normalize cross-correlation
    float alpha = static_cast<float>(template_img2_sum) -
        static_cast<float>(template_img_sum * template_img_sum) / (TM * TN * TK);
    alpha = alpha <= 0.0f ? 0.0f : 1 / sqrtf(alpha);
    err = hipStreamSynchronize(s1);
    if (hipSuccess != err) {
        printf("Fail to compute sum table, CUDA error %d!\n", err);
        return -1;
    }
    err = hipStreamSynchronize(s2);
    if (hipSuccess != err) {
        printf("Fail to compute sum table, CUDA error %d!\n", err);
        return -1;
    }
    normalize3DCC_kernel<T,
        Layout3D<Shape3D<OM, ON, OK>, Stride3D<1, 256, 256 * 256>>,
        Layout3D<Shape3D<IM, IN, IK>>,
        Layout3D<Shape3D<OM, ON, OK>, Stride3D<1, 256, 256 * 256>>> << <(OM * ON * OK + 1024 - 1) / 1024, 1024 >> > (dconv, dimg_sumtable, energy_sumtable, template_img_sum, static_cast<T>(alpha), doutput);

    hipEventRecord(end);
    hipEventSynchronize(end);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, beg, end);
    printf("elapsed time = %.2fms.\n", elapsedTime);
    hipEventDestroy(beg);
    hipEventDestroy(end);

    // get NCC result
    std::vector<T> output(OM * ON * OK);
    for (int k = 0; k < OK; k++) {
        for (int j = 0; j < ON; j++) {
            hipMemcpy(output.data() + j * OM + k * OM * ON, doutput + j * 256 + k * 256 * 256,
                sizeof(T) * OM, hipMemcpyDeviceToHost);
        }
    }
    auto cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        printf("Fail to comput 3D NCC! CUDA error: %d\n", cuda_error);
        return -1;
    }

    // find the max val in 3D NCC map
    T* d_max_val;
    int* d_max_idx;
    hipMalloc(&d_max_val, ALIGN128(sizeof(T) * OM * ON * OK));
    hipMalloc(&d_max_idx, ALIGN128(sizeof(int) * OM * ON * OK));

    find_minmax_16384_kernel<T, Layout3D<Shape3D<OM, ON, OK>, Stride3D<1, 256, 256 * 256>>, GT<T>> << <(OM * ON * OK + 16383) / 16384, 1024 >> > (doutput, d_max_val, d_max_idx);

    std::vector<T> h_max_val(((OM * ON * OK + 16383) / 16384));
    std::vector<int> h_max_idx(((OM * ON * OK + 16383) / 16384));
    hipMemcpy(h_max_val.data(), d_max_val, sizeof(T) * h_max_val.size(), hipMemcpyDeviceToHost);
    hipMemcpy(h_max_idx.data(), d_max_idx, sizeof(int) * h_max_idx.size(), hipMemcpyDeviceToHost);

    cuda_error = hipGetLastError();
    if (hipSuccess != cuda_error) {
        printf("Fail to find max val, CUDA error %d!\n", cuda_error);
        return -2;
    }
    T max_val = GT<T>::dummy;
    int max_idx = 0;
    for (int i = 0; i < h_max_val.size(); i++) {
        if (GT<T>::cmp(h_max_val[i], max_val)) {
            max_val = h_max_val[i];
            max_idx = h_max_idx[i];
        }
    }
    printf("NCC max val = %f, coordinate = (%d, %d, %d)\n", max_val,
        max_idx % OM, (max_idx / OM) % ON, max_idx / (OM * ON));

    // destroy CUDA streams
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipStreamDestroy(s3);
    hipStreamDestroy(s4);

    // output
    std::ofstream outs("ncc-65x65x65.raw", std::ios::binary);
    outs.write((char*)output.data(), sizeof(T) * output.size());
    outs.close();

    // release
    hipFree(dimage);
    hipFree(dimg_sumtable);
    hipFree(energy_sumtable);
    hipFree(d_template_img);
    hipFree(d_sum_workspace);
    hipFree(d_sum2_workspace);
    hipFree(dconv);
    hipFree(d_max_val);
    hipFree(d_max_idx);

    // over
    return 0;
}

namespace { // use this kernel only in current compilation-unit!
    __global__ void xcorr_kernel(hipComplex* X, hipComplex* Y,
        const int N, const float alpha, hipComplex* Z) {
        hipComplex x, y;
        const int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (N > idx) {
            x = X[idx];
            y = Y[idx];

            x.x *= alpha;
            x.y *= alpha;

            Z[idx] = hipCmulf(x, hipConjf(y));
        }
        // over
        return;
    }
}

// use cuFFT to perform 3D CC computation
template <typename Layout>
int conv3D_using_FFT(float* dX3D, float* dY3D, float* dZ3D) {
    // we use 3D FFT plans
    hipfftHandle plan3D_fft, plan3D_ifft;
    hipfftResult cufft_err;

    cufft_err = hipfftPlan3d(&plan3D_fft,
        Layout::Shape::ShapeZ, Layout::Shape::ShapeY, Layout::Shape::ShapeX, hipfftType::HIPFFT_R2C);
    if (HIPFFT_SUCCESS != cufft_err) {
        printf("Fail to make CUFFT plan! CUFFT error: %d\n", cufft_err);
        return -1;
    }

    cufft_err = hipfftPlan3d(&plan3D_ifft,
        Layout::Shape::ShapeZ, Layout::Shape::ShapeY, Layout::Shape::ShapeX, hipfftType::HIPFFT_C2R);
    if (HIPFFT_SUCCESS != cufft_err) {
        printf("Fail to make CUFFT plan for IFFT! CUFFT error: %d\n", cufft_err);
        return -2;
    }

    hipfftComplex* dFX3D, * dFY3D;
    const int fft_size = Layout::Shape::ShapeZ * Layout::Shape::ShapeY * (Layout::Shape::ShapeX / 2 + 1);
    hipMalloc(&dFX3D, ALIGN128(sizeof(hipfftComplex) * fft_size));
    hipMalloc(&dFY3D, ALIGN128(sizeof(hipfftComplex) * fft_size));
    // FFT
    cufft_err = hipfftExecR2C(plan3D_fft, dX3D, dFX3D);
    if (HIPFFT_SUCCESS != cufft_err) {
        printf("Fail to transform image! CUFFT error: %d\n", cufft_err);
        return -3;
    }
    cufft_err = hipfftExecR2C(plan3D_fft, dY3D, dFY3D);
    if (HIPFFT_SUCCESS != cufft_err) {
        printf("Fail to transform template! CUFFT error: %d\n", cufft_err);
        return -4;
    }
    // apply convolution theorem in frequency domain
    xcorr_kernel << <(fft_size + 511) / 512, 512 >> > (dFX3D, dFY3D,
        fft_size, 1.0f / static_cast<float>(Layout::size()), dFX3D);
    // IFFT to get result
    cufft_err = hipfftExecC2R(plan3D_ifft, dFX3D, dZ3D);
    if (HIPFFT_SUCCESS != cufft_err) {
        printf("Fail to compute IFFT! CUFFT error: %d\n", cufft_err);
        return -5;
    }

    hipDeviceSynchronize();
    auto cuda_err = hipGetLastError();
    if (hipSuccess != cuda_err) {
        printf("Fail to compute CC! CUDA error %d!\n", cuda_err);
        return -6;
    }
    // release
    hipFree(dFX3D);
    hipFree(dFY3D);

    hipfftDestroy(plan3D_fft);
    hipfftDestroy(plan3D_ifft);
    // over
    return 0;
}
